#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include "random_points.h"
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

// Function to compute distance between points
__device__ double distance(Point p1, Point p2) {
    double dx = p1.x - p2.x;
    double dy = p1.y - p2.y;
    return sqrt(dx * dx + dy * dy);
}

// Function to compute distance between a line PQ and a point
__device__ double distanceToLine(Point lineStart, Point lineEnd, Point point) {
    double lineLength = distance(lineStart, lineEnd);
    double area = 0.5f * fabsf(
        (lineEnd.x - lineStart.x) * (lineStart.y - point.y) - 
        (lineStart.x - point.x) * (lineEnd.y - lineStart.y)
    );
    return (2.0f * area) / lineLength;
}

// Check if a point is above the line PQ in a clockwise orientation
__device__ bool isAboveClockwise(Point p, Point q, Point point) {
	// Cross product gives which is the orientation
    double cross = ((q.x - p.x) * (point.y - p.y)) - ((point.x - p.x) * (q.y - p.y));
    return cross > 0;
}

// Kernel to compute the convex hull given a set of points in the plane
__global__ void quickHullKernel(Point* points, int numPoints, Point left, Point right, int* hullPoints, int* numHullPoints) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= numPoints) return;

    double maxDistance = -1.0f;
    int maxIndex = -1;

    // Find farthest point
    for (int i = tid; i < numPoints; i += blockDim.x * gridDim.x) {
    	if (isAboveClockwise(left, right,  points[i])) {
        	double distance = distanceToLine(left, right, points[i]);
        	if (distance > maxDistance) {
            	maxDistance = distance;
            	maxIndex = i;
            }
        }
    }

    // Add the resulting point to the hull
    if (maxIndex >= 0) {
        int index = atomicAdd(numHullPoints, 1);
        hullPoints[index] = maxIndex;

        Point p = points[maxIndex];

        // Find left-most points
        if (numPoints > 0) {
            int innerIndex = atomicAdd(numHullPoints, 1);
            hullPoints[innerIndex] = tid;
        }
    }
}


int main() {
	Point* points;
	int* hullPoints;
	int* numHullPoints;
	// Generate random points in the plane using the function from the generator
	vector<Point> r_points = generate_random_points();
	cout << "Points randomly generated!" << endl;
	
	// Let's use Unified Memory
	hipMallocManaged((void**)&points, N*sizeof(Point)); // input
	hipMallocManaged((void**)&hullPoints, N*sizeof(int)); // output
	hipMallocManaged(&numHullPoints, sizeof(int)); // output size
	*numHullPoints = 0; // first output size
	
	// From vector to array
	for (int i = 0; i < N; i++) {
    	points[i].x = r_points[i].x;
    	points[i].y = r_points[i].y;
	}
	
	Point left = points[0];
	Point right = points[0];
	// Find left-most and right-most points in the set
	for (int i = 1; i < N; i++) {
    	if (points[i].x < left.x)
        	left = points[i];
    	if (points[i].x > right.x)
        	right = points[i];
	}

	// Set Block and Grid dimension
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Timer starts
    auto start = chrono::high_resolution_clock::now();
    
	// Launch Kernels
	quickHullKernel<<<blocksPerGrid, threadsPerBlock>>>(points, N, left, right, hullPoints, numHullPoints);
	quickHullKernel<<<blocksPerGrid, threadsPerBlock>>>(points, N, right, left, hullPoints, numHullPoints);
	// synchronize
	hipDeviceSynchronize();
	
	// Timer stops
	auto end = chrono::high_resolution_clock::now();

	// Print first 3 elements
	for (int i = 0; i < 3; i++) {
    	printf("Hull Point %d: (%f, %f)\n", i, points[hullPoints[i]].x, points[hullPoints[i]].y);
	}
	
	cout << ".\n.\n." << endl;
	
	// Print last 3 elements
	for (int i = *numHullPoints-3; i < *numHullPoints; i++) {
    	printf("Hull Point %d: (%f, %f)\n", i, points[hullPoints[i]].x, points[hullPoints[i]].y);
	}

	// Compute time interval
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start).count();

    cout << "Execution time: " << duration << " ms" << endl;
	
	// Free memory space
	hipFree(points);
	hipFree(hullPoints);

	return 0;
}